#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.
// This file is modified from  https://github.com/pytorch/pytorch/blob/master/modules/detectron/sigmoid_focal_loss_op.cu
// Cheng-Yang Fu
// cyfu@cs.unc.edu
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <cfloat>

// TODO make it in a common file
#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)


template <typename T>
__global__ void SigmoidFocalLossForward(const int nthreads, 
    const T* logits,
    const int* targets,
    const int num_classes,
    const float gamma, 
    const float alpha,
    const int num, 
    T* losses) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {

    int n = i / num_classes;
    int d = i % num_classes; // current class[0~79]; 
    int t = targets[n]; // target class [1~80];

    // Decide it is positive or negative case. 
    T c1 = (t == (d+1)); 
    T c2 = (t>=0 & t != (d+1));

    T zn = (1.0 - alpha);
    T zp = (alpha);

    // p = 1. / 1. + expf(-x); p = sigmoid(x)
    T  p = 1. / (1. + expf(-logits[i]));

    // (1-p)**gamma * log(p) where
    T term1 = powf((1. - p), gamma) * logf(max(p, FLT_MIN));

    // p**gamma * log(1-p)
    T term2 = powf(p, gamma) *
            (-1. * logits[i] * (logits[i] >= 0) -   
             logf(1. + expf(logits[i] - 2. * logits[i] * (logits[i] >= 0))));

    losses[i] = 0.0;
    losses[i] += -c1 * term1 * zp;
    losses[i] += -c2 * term2 * zn;

  } // CUDA_1D_KERNEL_LOOP
} // SigmoidFocalLossForward


template <typename T>
__global__ void SigmoidFocalLossBackward(const int nthreads,
                const T* logits,
                const int* targets,
                const T* d_losses,
                const int num_classes,
                const float gamma,
                const float alpha,
                const int num,
                T* d_logits) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {

    int n = i / num_classes;
    int d = i % num_classes; // current class[0~79]; 
    int t = targets[n]; // target class [1~80], 0 is background;

    // Decide it is positive or negative case. 
    T c1 = (t == (d+1));
    T c2 = (t>=0 & t != (d+1));

    T zn = (1.0 - alpha);
    T zp = (alpha);
    // p = 1. / 1. + expf(-x); p = sigmoid(x)
    T  p = 1. / (1. + expf(-logits[i]));

    // (1-p)**g * (1 - p - g*p*log(p)
    T term1 = powf((1. - p), gamma) *
                      (1. - p - (p * gamma * logf(max(p, FLT_MIN))));

    // (p**g) * (g*(1-p)*log(1-p) - p)
    T term2 = powf(p, gamma) *
                  ((-1. * logits[i] * (logits[i] >= 0) -
                      logf(1. + expf(logits[i] - 2. * logits[i] * (logits[i] >= 0)))) *
                      (1. - p) * gamma - p);
    d_logits[i] = 0.0;
    d_logits[i] += -c1 * term1 * zp;
    d_logits[i] += -c2 * term2 * zn;
    d_logits[i] = d_logits[i] * d_losses[i];

  } // CUDA_1D_KERNEL_LOOP
} // SigmoidFocalLossBackward


at::Tensor SigmoidFocalLoss_forward_cuda(
		const at::Tensor& logits,
                const at::Tensor& targets,
		const int num_classes, 
		const float gamma, 
		const float alpha) {
  AT_ASSERTM(logits.type().is_cuda(), "logits must be a CUDA tensor");
  AT_ASSERTM(targets.type().is_cuda(), "targets must be a CUDA tensor");
  AT_ASSERTM(logits.dim() == 2, "logits should be NxClass");

  const int num_samples = logits.size(0);
	
  auto losses = at::empty({num_samples, logits.size(1)}, logits.options());
  auto losses_size = num_samples * logits.size(1);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(THCCeilDiv((long)losses_size, 512L), 4096L));
  
  dim3 block(512);

  if (losses.numel() == 0) {
    THCudaCheck(hipGetLastError());
    return losses;
  }

  AT_DISPATCH_FLOATING_TYPES(logits.type(), "SigmoidFocalLoss_forward", [&] {
    SigmoidFocalLossForward<scalar_t><<<grid, block, 0, stream>>>(
         losses_size,
         logits.contiguous().data<scalar_t>(),
	 targets.contiguous().data<int>(),
         num_classes,
	 gamma,
	 alpha,
	 num_samples,
         losses.data<scalar_t>());
  });
  THCudaCheck(hipGetLastError());
  return losses;   
}	


at::Tensor SigmoidFocalLoss_backward_cuda(
		const at::Tensor& logits,
                const at::Tensor& targets,
		const at::Tensor& d_losses,
		const int num_classes, 
		const float gamma, 
		const float alpha) {
  AT_ASSERTM(logits.type().is_cuda(), "logits must be a CUDA tensor");
  AT_ASSERTM(targets.type().is_cuda(), "targets must be a CUDA tensor");
  AT_ASSERTM(d_losses.type().is_cuda(), "d_losses must be a CUDA tensor");

  AT_ASSERTM(logits.dim() == 2, "logits should be NxClass");

  const int num_samples = logits.size(0);
  AT_ASSERTM(logits.size(1) == num_classes, "logits.size(1) should be num_classes");
	
  auto d_logits = at::zeros({num_samples, num_classes}, logits.options());
  auto d_logits_size = num_samples * logits.size(1);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(THCCeilDiv((long)d_logits_size, 512L), 4096L));
  dim3 block(512);

  if (d_logits.numel() == 0) {
    THCudaCheck(hipGetLastError());
    return d_logits;
  }

  AT_DISPATCH_FLOATING_TYPES(logits.type(), "SigmoidFocalLoss_backward", [&] {
    SigmoidFocalLossBackward<scalar_t><<<grid, block, 0, stream>>>(
         d_logits_size,
         logits.contiguous().data<scalar_t>(),
	 targets.contiguous().data<int>(),
	 d_losses.contiguous().data<scalar_t>(),
         num_classes,
	 gamma,
	 alpha,
	 num_samples,
         d_logits.data<scalar_t>());
  });

  THCudaCheck(hipGetLastError());
  return d_logits;   
}	

